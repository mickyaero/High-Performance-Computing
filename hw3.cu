//CUDA code for matrix multiplicationn . The values of a,b,c,q have to changed according to N
#include<stdlib.h>
#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
__global__ void Product (float *a, float *b, float *c)
{
// Out of all the threads created each one computes 1 value of C and stores into cval

float cval = 0.00;
int R = blockIdx.y * blockDim.y + threadIdx.y; //Row of the matrix
int C = blockIdx.x * blockDim.x + threadIdx.x; //Column of the matrix
//Defining the size of the matrix//
int N=1000;
if(R> N || C > N ){
    return;
}
for (int j = 0; j < N; j++)
{
cval += a[R * N+ j] *b[j * N + C];
			
}
c[R * N + C]+= cval;                     
}                       
                       
  using namespace std;                

int main(){

//The timing function         
hipEvent_t start,stop;
float time;	
int N=5000;	
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);	

static float a[25000000],b[25000000];
static float c[25000000];



//Inputting values in the matrix
        
long int  q = 25000000;   // Standard int runs out of memory so long int used
int i=0;
//For checking the matrix multiplication all entries are 1
 while(i != q)
 {
   a[i] = 1;
   b[i] = 1;
  
   i++;
                }

int o=0;
//for(int m=0;m<N;m++){
//for(int n=0;n<N;n++){
//a[o]=m+n;
//b[o]=m*n;
//o=o+1;
//}}
//This section is the GPU part

        float *device_a, *device_b, *device_c;
	dim3 griddimension(500,500); // The dimension of the total grid (Blocks)
	dim3 blockdimension(10,10);  // The dimension of one block ( threads in one block)

//Allocating memory in the device for the matrices: device_a,b,c are device variables
hipMalloc( (void**)&device_c, q * sizeof(float) );
hipMalloc( (void**)&device_b, q * sizeof(float) );
hipMalloc( (void**)&device_a, q * sizeof(float) );
//Copying the variables from CPU to GPU
hipMemcpy( device_a,a,q * sizeof(float),hipMemcpyHostToDevice );
hipMemcpy( device_b,b,q * sizeof(float),hipMemcpyHostToDevice );
hipMemcpy( device_c,c,q * sizeof(float),hipMemcpyHostToDevice );

Product<<<griddimension, blockdimension>>>( device_a, device_b, device_c ); //The device function Product is called

hipMemcpy( c,device_c,q * sizeof(float),hipMemcpyDeviceToHost );
hipFree( device_a );
hipFree( device_b );
hipFree( device_c );

hipEventRecord(stop,0);
hipEventSynchronize(stop);

hipEventElapsedTime(&time,start,stop);
cout<<"\n\nTime = "<<time<<" ms";

//For printing the matrix
long int g=N*N,d=0;
while(d!=g){
printf("%f\n",c[d]);
d=d+1;
}
//}


}
              
