#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <iomanip>
#define n 1000

__global__ void Matrix_Product (double *A, double *g, double *C)
// Each thread computes one element of C
// by accumulating results into Cvalue 
{               double Cvalue = 0.00;
                int row = blockIdx.y*blockDim.y+threadIdx.y;
               // int col = blockIdx.x * blockDim.x + threadIdx.x;
        //size of matrix A//
                int N=1000;
                if(row> N ) return;
                for (int e = 0; e < N; e++)
                        {
                        Cvalue += A[N*row+e]*g[e];
                        }
                 C[row]+= Cvalue;                     
}

using namespace std;
int main(){

double a[n*n],x[n],c[n],temp=0,d=2;
 
srand(time(NULL));
for(long int i=0;i<n*n;i++)
{
		a[i]=2*i*314.9568298+100;	
		//cin>>a[i][j];                //generating the matrix a[n][n]
		//cout<<" "<<a[i][j]<<endl;
}
//
for(int i=0;i<n;i++)
{
	x[i]=0.5;
}
x[n-1]=1;

hipEvent_t start,stop;
        float elapsedTime;    

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0); 

	double *dev_a, *dev_x, *dev_c;
        dim3 griddim(100,1);
        dim3 blockdim(10,1);
	hipMalloc( (void**)&dev_a, n *n* sizeof(double) );
	 hipMalloc( (void**)&dev_c, n * sizeof(double) );
        hipMalloc( (void**)&dev_x, n * sizeof(double) );
	 hipMemcpy( dev_a,a,n * n * sizeof(double),hipMemcpyHostToDevice );
	

 while(fabs(d-temp)>0.0000000000001)
    {
		
        for(int i=0;i<n;i++)
        {
            c[i]=0;
	}
           // for(int j=0;j<n;j++)        //portion to be parallelized
	//		{
          //     			 c[i]+=a[i][j]*x[j];
	//		}


  //      cudaMalloc( (void**)&dev_c, n * sizeof(double) );
//        cudaMalloc( (void**)&dev_x, n * sizeof(double) );
//        cudaMalloc( (void**)&dev_a, n *n* sizeof(double) );

        //cudaMemcpy( dev_a,a,n * n * sizeof(double),cudaMemcpyHostToDevice );
        hipMemcpy( dev_x,x,n * sizeof(double),hipMemcpyHostToDevice );
        hipMemcpy( dev_c,c,n * sizeof(double),hipMemcpyHostToDevice );

        Matrix_Product<<<griddim, blockdim>>>( dev_a, dev_x, dev_c );

        hipMemcpy( c,dev_c,n * sizeof(double),hipMemcpyDeviceToHost );

//        cudaFree( dev_a );
  //      cudaFree( dev_x );
    //    cudaFree( dev_c );

        
        for(int i=0;i<n;i++)
		{
        	    x[i]=c[i];
		}
        temp=d;
        d=0;
        
        for(int i=0;i<n;i++)
        {
            if(fabs(x[i])>fabs(d))
                d=x[i];
        }
        for(int i=0;i<n;i++){
            x[i]/=d;
		}
    }
//	 cudaMemcpy( c,dev_c,n * sizeof(double),cudaMemcpyDeviceToHost );


	   hipFree( dev_a );
        hipFree( dev_x );
        hipFree( dev_c );
    
 hipEventRecord(stop,0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsedTime,start,stop);
       cout<<"\n\nElapsed Time = "<<elapsedTime<<" ms";

    //cout<<d<<endl;
    //for(int i=0;i<n;i++){
	//	cout<<setprecision(30)<<d<<endl;
	//}
//cout<<"Enter the initial guess for eigen vector";
//for(int i=0;i<n;i++){
//	cout<<x[i]<<endl;
//}
//}

return 0;
}
