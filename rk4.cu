
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

using namespace std;

__host__ __device__ void deriv(double* x0, double* array, double c, int size,  double* ki, double*
        ko);
__global__ void rk4(double* x0, double* array, double* h, int size);

int main(){

    int n_voxel = 10;
    int n_species = 100;
    int species_square = n_species * n_species;
    double x0[n_species];
    double a_mat[species_square];
    double h = 0.005;
    
    double duration = 0.0;
    //Filling up the a_mat and x0
    
    for(int i = 0; i < n_species; ++i){
        x0[i] = i;
        for(int j = 0; j < n_species; ++j){
            a_mat[i * n_species + j] = rand() % 9 + 1; 
        }
    }

    
    //Porting the problem onto gpu
    double *d_x0, *d_a_mat, d_h;
    int  d_n_species;
    
    clock_t start = clock();

    hipMalloc( (void**)&d_x0, sizeof(double) * n_species );
    hipMalloc( (void**)&d_a_mat, sizeof(double) * species_square );
    hipMalloc( (void**)&d_h, sizeof(double) * 1 );
    hipMalloc( (void**)&d_n_species, sizeof(int) * 1 );

    hipMemcpy( d_x0, x0, sizeof(double) * n_species, hipMemcpyHostToDevice );
    hipMemcpy( d_a_mat, a_mat, sizeof(double) * species_square, hipMemcpyHostToDevice );
    hipMemcpy( &d_h, &h, sizeof(double) * 1, hipMemcpyHostToDevice );
    hipMemcpy( &d_n_species, &n_species, sizeof(int) * 1, hipMemcpyHostToDevice );

    dim3 blocks( 1, 1, 1 );
    dim3 threads( 10, 1, 1 );

    rk4 <<< blocks, threads >>> ( d_x0, d_a_mat, &d_h, d_n_species );
    
    hipMemcpy( x0, d_x0, sizeof(double) * n_species, hipMemcpyDeviceToHost );
    hipMemcpy( a_mat, d_a_mat, sizeof(double) * species_square, hipMemcpyDeviceToHost );
    
    hipFree( d_x0 );
    hipFree( d_a_mat );
    hipFree( &d_h );
    hipFree( &d_n_species );

    duration = static_cast<double>(clock() - start);
    cout << "Time = " << duration << endl;

/**
    //Running rk4 over all the voxels
    clock_t start = clock();
    for(int i = 0; i < n_voxel; ++i){
        rk4(x0, a_mat, &h, n_species);
       // cout << "x0 address = " << x0 << endl;
       // cout << "x0 value = " << x0[0] << endl;
   }
   duration = static_cast<double>(clock() - start);
   cout << "Time = " << duration << endl;

   **/
}

__host__ __device__ void deriv(double* x0, double* array, double c, int size,  double* ki, double* ko){

    for(int i = 0; i < size; ++i){

        for(int j = 0; j< size; ++j){

            ko[i] = ko[i] + array[ size * i + j ] * (x0[j] + c * ki[j]); 

        }

    }

}

__global__ void rk4(double* x0, double* array, double* h, int size){
    //int** arr = new int*[row];
    //int size = 100;//size of the species
    double* k1_v = new double [size];
    double* k2_v = new double [size];
    double* k3_v = new double [size];
    double* k4_v = new double [size];
    for(int i = 0; i < size; ++i){

        k1_v[i] = x0[i];
        k2_v[i] = x0[i];
        k3_v[i] = x0[i];
        k4_v[i] = x0[i];
    
    }

    deriv(x0, array, 0.0, size, x0, k1_v);
    deriv(x0, array, *h/2.0, size, k1_v, k2_v);
    deriv(x0, array, *h/2.0, size, k2_v, k3_v);
    deriv(x0, array, *h, size, k3_v, k4_v);

    for(int i = 0; i < size; ++i){
        x0[i] = x0[i] + (k1_v[i] + 2.0 *  k2_v[i] + 2.0 * k3_v[i] + k4_v[i]) *
            (*h)/6.0;  
    }
    delete[] k1_v;
    delete[] k2_v;
    delete[] k3_v;
    delete[] k4_v;
   // delete[] arr;
}


